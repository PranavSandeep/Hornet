#include "hip/hip_runtime.h"
#include "pch.h"
#include <Windows.h>
#include <string>
#include <cmath>

__global__ void cudaAddVectors(float* a, float* b, float* result, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        result[idx] = a[idx] + b[idx];
    }
}

__global__ void cudaMultiplyVectors(float* a, float* b, float* result, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        result[idx] = a[idx] * b[idx];
    }
}

extern "C" void addVectors(float* a, float* b, float* result, int size) {
    // Allocate device memory
    float* d_a, * d_b, * d_result;
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));
    hipMalloc((void**)&d_result, size * sizeof(float));

    // Copy input vectors to device
    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    cudaAddVectors << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_result, size);

    // Copy result back to host
    hipMemcpy(result, d_result, size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}

extern "C" float dot(float* a, float* b, float* result, int size)
{
    float* d_a, * d_b, * d_result;
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));
    hipMalloc((void**)&d_result, size * sizeof(float));

    //Copy input vectors to device
    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    cudaMultiplyVectors << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_result, size);

    // Copy result back to host
    hipMemcpy(result, d_result, size * sizeof(float), hipMemcpyDeviceToHost);

    float s = 0.f;

    for (int i = 0; i < size; i++)
    {
        s += result[i];
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return s;
}

extern "C" float magnitude(float* a, float* result, const int size)
{
    float s = 0.f;
    s = dot(a, a, result, size);

    return sqrt(s);
}




extern "C"
{

	__declspec(dllexport) int add(int a, int b)
	{
		return a + b;
	}

	__declspec(dllexport) void add_vec(float* a, float* b, float* result, int size)
	{
		addVectors(a, b, result, size);
	}

    __declspec(dllexport) float cdot(float* a, float* b,  float* result, int size)
    {
        float d = 0.f;
        d = dot(a, b, result, size);

        return d;
    }

    __declspec(dllexport) float cmagnitude(float* a, float* result, int size)
    {
        float m = 0.f;
        m = magnitude(a, result, size);

        return m;
    }
};